
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_SIZE 1024

__global__ void cu_dotProd (long *array, long arraySize)
{
    long x = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    if (x < arraySize){
      long f = 0;
      long d = 0;

      if (x < arraySize/2){
         f = x + 1;
      } else {
        f = arraySize - x;
      }

      if (x%10 == 0) {
        d = 1;
      } else {
        d = (x%10) + 1;
      }
      array[x] = f * d;
    }
    //__syncthreads();
}

extern "C" void dotProduct(long *arr, long arraySize)
{
  long *arrayR_d;
	hipError_t result;

	// allocate space in the device for arrayR_d
	result = hipMalloc ((void**) &arrayR_d, sizeof(long) * arraySize);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMalloc failed.");
		exit(1);
	}

  // printf ("dimgrid:\n");
  // printf ("%d %d %f ", arraySize, BLOCK_SIZE, ceil((float)arraySize/BLOCK_SIZE));
  // printf ("\n");

	// set execution configuration
	dim3 dimblock (BLOCK_SIZE);
	//dim3 dimgrid (arraySize/BLOCK_SIZE);
  dim3 dimgrid(ceil((float)arraySize/BLOCK_SIZE));

	// actual computation: Call the kernel
	cu_dotProd <<<dimgrid, dimblock>>> (arrayR_d, arraySize);

	// transfer result back to host
	result = hipMemcpy (arr, arrayR_d, sizeof(long) * arraySize, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed.");
		exit(1);
	}

 //  printf ("Final state of the array arrayR_d:\n");
 //  int j;
 //  for (j=0; j < arraySize; j++) {
 //     //printf ("%d ", j);
 //     printf ("%d ", arr[j]);
 //  }
 // printf ("\n");

	result = hipFree (arrayR_d);
	if (result != hipSuccess) {
		fprintf(stderr, "hipFree failed.");
		exit(1);
	}
}
